#include "hip/hip_runtime.h"
/// Accelerating Spike Propagation for GPU-based Spiking Neural Network Simulations

/// The reference code is taken from the state-of-the-art (SOTA) implementataion: 
/// SOTA : "Dynamic parallelism for synaptic updating in GPU-accelerated spiking neural network simulations"
/// Paper: https://www.sciencedirect.com/science/article/pii/S0925231218304168
/// Code: https://bitbucket.org/bkasap/dynamicparallelismsnn/src/master/

/// This repository includes SOTA (AP, N, S) algorithms and modified (AB, SKL) algorithms.

#include <unistd.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <fstream>
#include <cstdio>
#include <cstring>
#include <sstream>
#include <ctime>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <sys/stat.h>

#include <assert.h>
///#include <stdio.h>
///#include <stdlib.h>
#include <string.h>
//#include <sm_11_atomic_functions.h>

#include <hip/hip_runtime.h>

// Utilities and system includes
#include <hip/hip_runtime_api.h>  // helper function CUDA error checking and initialization
#include <helper_functions.h>  // helper for shared functions common to CUDA Samples

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
using namespace cooperative_groups; 

////////////////

using namespace std;

class Neuron
{
	public:
		float	v;
		float 	u;
		float	a;
		float	b;
		float	c;
		float	d;
		float	I;
		int		nospks;
		int		neuronid;
};

////////////////////////////// global sync here ///////////////////////////

//#define ITER_COUNT 2000

// Initialize the random states
__global__ void randgeneratorinit(unsigned int seed, hiprandState_t* states, int N) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if ( id < N ){
	/* we have to initialize the state */
		hiprand_init(seed,	/* the seed can be the same for each core, here we pass the time in from the CPU */
			  id,		/* the sequence number should be different for each core (unless you want all
							 cores to get the same sequence of numbers for some reason - use thread id! */
			  0,		/* the offset is how much extra we advance in the sequence for each call, can be 0 */
			  &states[id]);
		}
}

// Initialize the neural parameters
__global__ void initNeuron(int N_exc, int N, Neuron *neuron, hiprandState_t* state){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	//printf("%d \n", id);

	hiprandState_t localState = state[id];
	float rand_float = hiprand_uniform(&localState);
	// initialize excitatory neuron parameters
	if ( id < N ){
	if ( id < N_exc ){
		neuron[id].a = 0.02f;
		neuron[id].b = 0.2f;
		neuron[id].c = -65.0f+(15.f*powf(rand_float,2.0f));
		neuron[id].d = 8.0f-(6.f*powf(rand_float,2.0f));
		neuron[id].v = -65.0f;
		neuron[id].u = neuron[id].v*neuron[id].b;
		neuron[id].I = 0.0f;
		neuron[id].nospks = 0;
		neuron[id].neuronid = id;
	}
	// initialize inhibitory neuron parameters
	else if (id >= N_exc and id < N ){
		neuron[id].a = 0.02f+(0.08f*rand_float);
		neuron[id].b = 0.25f-(0.05f*rand_float);
		neuron[id].c = -65.0f;
		neuron[id].d = 2.0f;
		neuron[id].v = -65.0f;
		neuron[id].u = neuron[id].v*neuron[id].b;
		neuron[id].I = 0.0f;
		neuron[id].nospks = 0;
		neuron[id].neuronid = id;
	}

	state[id] = localState;
	}
}

// Propagate spikes with dynamic parallelization: AP-algorithm
__global__ void propagatespikes(int spiked, float *d_conn_w, int *d_conn_idx, float *d_Isyn, const int N, const int N_syn){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	int dst_idx;
	//AP-Algorithm
	if ( id < N_syn and spiked < N ){
		dst_idx = (int)spiked*N_syn+id;
		atomicAdd(&d_Isyn[d_conn_idx[dst_idx]], d_conn_w[dst_idx]);
	}
}

// State update of neural variables
__global__ void stateupdate(Neuron *neuron,			// Neural parameters of individual neurons
							bool *spike,			// List of booleans to keep spiking neuron indices
							float *d_conn_w,		// Connectivity strengths
							int *d_conn_idx,		// Connection target neuron ids
							float *d_Isyn,			// Synaptic inputs
							hiprandState_t *state,	// States for random number generators (RNG)
							const int N,//			// Number of neurons
							const int N_exc,		// Number of excitatory neurons
							//const int N_inh,		// Number of inhibitory neurons
							const int N_syn,		// Number of synapses per neuron
							int gridSize,			// Grid size and block size for the child kernels.
							int blockSize,
							int dynamic,
							int mode){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	//if (id==2499) printf(" id = %d %d %d %d", id, threadIdx.x, blockIdx.x, blockDim.x);
	//printf(" id = %d", threadIdx.x);
	
      if ( id < N ){
	if (!isfinite(neuron[id].I))
		neuron[id].I = 0.f;

	// Stochastic input current
	hiprandState_t localState = state[id];
	float rand_float = hiprand_normal(&localState);	

		// Selecting the conductances for varying firing regimes
		float ge, gi;
		switch ( mode ){
		case 0: // quiet regime
			ge = 2.5f; gi = 1.0f;
			break;
		case 1: // balanced regime
			ge = 5.0f; gi = 2.0f;
			break;
		case 2: // irregular regime
			ge = 7.5f; gi = 3.0f;
			//ge = 15.0f; gi = 6.0f;
			//ge = 30.0f; gi = 12.0f;
			break;
		}

		if ( id < N_exc ){
			neuron[id].I = ge*rand_float;		//5.0 for balanced regime, 7.5 for irregular, 2.5 for quiet
		}
		//else if (id >= N_exc and id < N )
		else {
			neuron[id].I = gi*rand_float;		//2.0 for balanced regime, 3.0 for irregular, 1.0 for quiet
		}

		// Current each neuron receives at a timestep
		// sum of the stochastic and synaptic inputs
		neuron[id].I += d_Isyn[id];

		// update state variables
		neuron[id].v += 0.5f*(0.04f*neuron[id].v*neuron[id].v + 5.0f*neuron[id].v + 140.f - neuron[id].u + neuron[id].I);
		neuron[id].v += 0.5f*(0.04f*neuron[id].v*neuron[id].v + 5.0f*neuron[id].v + 140.f - neuron[id].u + neuron[id].I);
		//neuron[id].v += 1.0f*(0.04f*neuron[id].v*neuron[id].v + 5.0f*neuron[id].v + 140.f - neuron[id].u + neuron[id].I);
		
		neuron[id].u += neuron[id].a*(neuron[id].b*neuron[id].v-neuron[id].u);

		// initialize currents for the next step already
		d_Isyn[id] = 0.f;

		// check if any neuron's membrane potential passed the spiking threshold
		//printf(" v value = %f ", neuron[id].v);
		if ( neuron[id].v >= 30.f or !isfinite(neuron[id].v) ){
			//printf(" hello ");
			spike[id] = true;
			neuron[id].v = neuron[id].c;
			neuron[id].u += neuron[id].d;
			neuron[id].nospks ++;
			// AP-algorithm for spike propagation
			// dynamic: 0 AP-algorithm
			if ( dynamic == 0 )
				propagatespikes<<<gridSize, blockSize>>>(id, d_conn_w, d_conn_idx, d_Isyn, N, N_syn);

		}
		else{
			spike[id] = false;
		}
		state[id] = localState;
	}	
}

// dynamic: 1  N-algorithm
__global__ void deliverspks1(Neuron *neuron,		// Neural parameters of individual neurons
							bool *spike,			// List of booleans to keep spiking neuron indices
							float *d_conn_w,		// Connectivity strengths
							int *d_conn_idx,		// Connection target neuron ids
							float *d_Isyn,			// Synaptic inputs
							const int N,			// Number of neurons
							const int N_exc,		// Number of excitatory neurons
							const int N_inh,		// Number of inhibitory neurons
							const int N_syn){		// Number of synapses per neuron
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	// N-algorithm
	// a thread for each neuron (presynaptic)
	if (id < N){
		// for each presynaptic neuron
		for (int dst_idx=0; dst_idx<N_syn; dst_idx++){
			// check if there is a spike from presynaptic neurons
			//d_conn_idx[id*N_syn+dst_idx];
			if (spike[id] == true){
				atomicAdd(&d_Isyn[d_conn_idx[id*N_syn+dst_idx]], d_conn_w[id*N_syn+dst_idx]);
			}
		}
	}
}

// dynamic: 2 S-algorithm
__global__ void deliverspks2(Neuron *neuron,		// Neural parameters of individual neurons
							bool *spike,			// List of booleans to keep spiking neuron indices
							float *d_conn_w,		// Connectivity strengths
							int *d_conn_idx,		// Connection target neuron ids
							float *d_Isyn,			// Synaptic inputs
							const int N,			// Number of neurons
							const int N_exc,		// Number of excitatory neurons
							const int N_inh,		// Number of inhibitory neurons
							const int N_syn){		// Number of synapses per neuron
	int id = threadIdx.x + blockIdx.x * blockDim.x;

	//int src;
	// S-algorithm
	// a thread for each synapse
	if (id < N*N_syn){
		//int src = (int) (id/N_syn);
		// check if there is a spike from presynaptic neurons
		if (spike[(id/N_syn)] == true){
			atomicAdd(&d_Isyn[d_conn_idx[id]], d_conn_w[id]);
		}
	}
}

////////////dynamic ==3   AB-algo stateupdate
__global__ void AB_stateupdate(Neuron *neuron,			// Neural parameters of individual neurons
							bool *spike,			// List of booleans to keep spiking neuron indices
							float *d_Isyn,			// Synaptic inputs
							hiprandState_t *state,	// States for random number generators (RNG)
							const int N,//			// Number of neurons
							const int N_exc,		// Number of excitatory neurons
							int mode){
     int id = threadIdx.x + blockIdx.x * blockDim.x;
	
     //grid_group grid = this_grid(); 
     
     while(id < N){
	  //id = threadIdx.x + block_id * blockDim.x;
	  //id = id + block_id;
	 // if ( id < N )
	 {

	  if (!isfinite(neuron[id].I))
		neuron[id].I = 0.f;

	  // Stochastic input current
	  hiprandState_t localState = state[id];
	  float rand_float = hiprand_normal(&localState);
	  //
	
		// Selecting the conductances for varying firing regimes
		float ge, gi;
		switch ( mode ){
		case 0: // quiet regime
			ge = 2.5f; gi = 1.0f;
			break;
		case 1: // balanced regime
			ge = 5.0f; gi = 2.0f;
			break;
		case 2: // irregular regime
			ge = 7.5f; gi = 3.0f;
			//ge = 15.0f; gi = 6.0f;
			//ge = 30.0f; gi = 12.0f;
			break;
		}

		if ( id < N_exc ){
			neuron[id].I = ge*rand_float;		//5.0 for balanced regime, 7.5 for irregular, 2.5 for quiet
		}
		//else if (id >= N_exc and id < N ){
		else{
			neuron[id].I = gi*rand_float;		//2.0 for balanced regime, 3.0 for irregular, 1.0 for quiet
		}

		// Current each neuron receives at a timestep
		// sum of the stochastic and synaptic inputs
		neuron[id].I += d_Isyn[id];
			
		// update state variables
		neuron[id].v += 0.5f*(0.04f*neuron[id].v*neuron[id].v + 5.0f*neuron[id].v + 140.f - neuron[id].u + neuron[id].I);
		neuron[id].v += 0.5f*(0.04f*neuron[id].v*neuron[id].v + 5.0f*neuron[id].v + 140.f - neuron[id].u + neuron[id].I);
		neuron[id].u += neuron[id].a*(neuron[id].b*neuron[id].v-neuron[id].u);

		// initialize currents for the next step already
		d_Isyn[id] = 0.f;
		//printf("d_count =  \t" );
		// check if any neuron's membrane potential passed the spiking threshold
		//printf(" v value = %f ", neuron[id].v);
		if ( neuron[id].v >= 30.f or !isfinite(neuron[id].v) ){
			//printf(" hello ");
			spike[id] = true;
			neuron[id].v = neuron[id].c;
			neuron[id].u += neuron[id].d;
			neuron[id].nospks ++;
			//printf("hello\t ");
		}
		else{
			spike[id] = false;   
		}
		state[id] = localState;
	  }
		id  = id + gridDim.x*blockDim.x;
	  
	 // block_id = block_id + gridDim.x;	
	}//while(id < N);	
}


// //dynamic ==3    AB-algorithm
__global__ void AB_deliverspks(Neuron *neuron,		// Neural parameters of individual neurons
							bool *spike,			// List of booleans to keep spiking neuron indices
							float *d_conn_w,		// Connectivity strengths
							int *d_conn_idx,		// Connection target neuron ids
							float *d_Isyn,			// Synaptic inputs
							const int N,			// Number of neurons
							const int N_exc,		// Number of excitatory neurons
							const int N_inh,		// Number of inhibitory neurons
							const int N_syn		// Number of synapses per neuron){
							){
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	//int id;
	//int block_id = blockIdx.x;
	//for(int id = id1; id < N*N_syn ; id  =+  gridDim.x*blockDim.x)
	//printf("gridSize=%d gridSize/\t", gridSize);
	while(id < N*N_syn )
	      {
		//id = threadIdx.x + block_id * blockDim.x;
		//int src;
		if (id < N*N_syn)
		{
			//int src = (int) (id/N_syn);
			// check if there is a spike from presynaptic neurons
			if ( spike[(id/N_syn)] == true){
				atomicAdd(&d_Isyn[d_conn_idx[id]], d_conn_w[id]);
			}
		}		
		id = id + gridDim.x*blockDim.x;
	     }	    
	//}while(block_id < gridSize );
}

/////dynamic ==4    under SKL-algorithm  SKL_deliverspks
// State update of neural variables
__device__ void SKL_stateupdate(Neuron *neuron,			// Neural parameters of individual neurons
							bool *spike,			// List of booleans to keep spiking neuron indices
							float *d_Isyn,			// Synaptic inputs
							hiprandState_t *state,	// States for random number generators (RNG)
							const int N,//			// Number of neurons
							const int N_exc,		// Number of excitatory neurons
							const int mode, int id){    //, int gridSizeN   
     
     while(id < N){
	  //id = threadIdx.x + block_id * blockDim.x;
	  //id = id + block_id;
	  //if ( id < N )
	 {

	  if (!isfinite(neuron[id].I))
		neuron[id].I = 0.f;

	  // Stochastic input current
	  hiprandState_t localState = state[id];
	  float rand_float = hiprand_normal(&localState);
	  //
	
		// Selecting the conductances for varying firing regimes
		float ge, gi;
		switch ( mode ){
		case 0: // quiet regime
			ge = 2.5f; gi = 1.0f;
			break;
		case 1: // balanced regime
			ge = 5.0f; gi = 2.0f;
			break;
		case 2: // irregular regime
			ge = 7.5f; gi = 3.0f;
			//ge = 15.0f; gi = 6.0f;
			//ge = 30.0f; gi = 12.0f;
			break;
		}

		if ( id < N_exc ){
			neuron[id].I = ge*rand_float;		//5.0 for balanced regime, 7.5 for irregular, 2.5 for quiet
		}
		//else if (id >= N_exc and id < N ){
		else {
			neuron[id].I = gi*rand_float;		//2.0 for balanced regime, 3.0 for irregular, 1.0 for quiet
		}

		// Current each neuron receives at a timestep
		// sum of the stochastic and synaptic inputs
		neuron[id].I += d_Isyn[id];
			
		// update state variables
		neuron[id].v += 0.5f*(0.04f*neuron[id].v*neuron[id].v + 5.0f*neuron[id].v + 140.f - neuron[id].u + neuron[id].I);
		neuron[id].v += 0.5f*(0.04f*neuron[id].v*neuron[id].v + 5.0f*neuron[id].v + 140.f - neuron[id].u + neuron[id].I);
		neuron[id].u += neuron[id].a*(neuron[id].b*neuron[id].v-neuron[id].u);

		// initialize currents for the next step already
		d_Isyn[id] = 0.f;
		//printf("d_count =  \t" );
		// check if any neuron's membrane potential passed the spiking threshold
		//printf(" v value = %f ", neuron[id].v);
		if ( neuron[id].v >= 30.f or !isfinite(neuron[id].v) ){
			//printf(" hello ");
			spike[id] = true;
			neuron[id].v = neuron[id].c;
			neuron[id].u += neuron[id].d;
			neuron[id].nospks ++;
			//printf("hello\t ");
		}
		else{
			spike[id] = false;   
		}
		state[id] = localState;
	  }
		id  = id + gridDim.x*blockDim.x;
	  
	 // block_id = block_id + gridDim.x;	
	}//while(id < N);		
   
}

// //dynamic ==4    SKL-algorithm
__global__ void SKL_deliverspks(Neuron *neuron,		// Neural parameters of individual neurons
							bool *spike,			// List of booleans to keep spiking neuron indices
							float *d_conn_w,		// Connectivity strengths
							int *d_conn_idx,		// Connection target neuron ids
							float *d_Isyn,			// Synaptic inputs
							const int N,			// Number of neurons
							const int N_exc,		// Number of excitatory neurons
							const int N_syn,		// Number of synapses per neuron
							//int *d_spkcount, 
							int *d_totalspkcount, 			// 
							hiprandState_t *state,
							const int mode, int d_count1){
	grid_group grid = this_grid();	 
	int d_count = d_count1;
	*d_totalspkcount = 0; 
	while(d_count){
	  d_count = d_count - 1;	
	  int id = threadIdx.x + blockIdx.x * blockDim.x;
	  
	  SKL_stateupdate(neuron, spike, d_Isyn, state, N, N_exc, mode, id);
	  grid.sync();
	  
	  id = threadIdx.x + blockIdx.x * blockDim.x;
	  while(id < N*N_syn){
		//id = threadIdx.x + block_id * blockDim.x;
		//int src;
		if (id < N*N_syn)
		{
			//int src = (int) (id/N_syn);
			// check if there is a spike from presynaptic neurons
			if ( spike[(id/N_syn)] == true){
				atomicAdd(&d_Isyn[d_conn_idx[id]], d_conn_w[id]);
			}			
		}		
		id = id + gridDim.x*blockDim.x;
	     }
	  id = threadIdx.x + blockIdx.x * blockDim.x; 	  	
	  grid.sync();
	  //  spike_count(spike, d_spkcount, N);
     	  while(id<N){
		if (  spike[id] == true ){
			//atomicAdd((int *)&d_spkcount[0], 1);	
			atomicAdd((int *)&d_totalspkcount[0], 1);						
		}
		id = id + gridDim.x*blockDim.x;	
	    }
	}//dcount
}


// Write filename
// helper functions to write simulation results in files
const char * filename(char * buffer, string varname){
	string fname;
	fname = buffer + varname;
	return fname.c_str();
}

void writeNeuronInfo(std::ofstream& output, Neuron n){
	output << n.a << " " << n.b << " " << n.c << " " << n.d << " " << n.v << " " << n.u << " " << n.I << " " << n.neuronid << " " << n.nospks << "\n";
}

// main function simulates the same network (N and S)
// for different firing regimes (mode: 0 quiet, 1 balanced, 2 irregular)
// with all the synaptic update algorithms (dynamic: 0 AP-algorithm, 1 N-algorithm, 2 S-algorithm)

int main(int argc, char **argv){
	// Number of neurons in the network: N
	// Number of synapses per neuron: N_syn
	const int N = atoi(argv[1]);
	const int N_syn = atoi(argv[2]);
	
	// (void *)&d_Neuron,  (void *)&d_spikes, (void *)&d_conn_w, (void *)&d_conn_idx, (void *)&d_Isyn, (void *)&N, (void *)&N_exc, (void *)&N_syn, (void *)&d_totalspkcount, (void *)&devStates, (void *)&mode, (void *)&d_count,};    				

	// Number of excitatory 80% and inhibitory 20% connections
	const int N_exc = ceil(4*N/5);
	const int N_inh = ceil(1*N/5);

	cout << "Number of neurons: " << N << "\nnumber of synapses per neuron: "  << N_syn << "\n";
	printf("%d Neurons in the network: %d excitatory and %d inhibitory \n", N, N_exc, N_inh);

	printf("Allocating space on GPU memory: \n");
	// Allocate space on GPU memory for neural parameters
	// for N neurons at time-steps t_n and t_(n+1)
	Neuron *d_Neuron, *h_Neuron;
	h_Neuron = (Neuron *)malloc(N*sizeof(Neuron));
	hipGetErrorString(hipMalloc(&d_Neuron, N*sizeof(Neuron)));

	
	// for N neurons to keep spikes
	bool *d_spikes, *h_spikes;
	h_spikes = (bool *)malloc(N*sizeof(bool));
	hipGetErrorString(hipMalloc(&d_spikes, N*sizeof(bool)));
	printf("Memory allocated for neurons\n");

	// for connectivity matrix
	float *d_conn_w; float *h_conn_w;
	h_conn_w = (float *)malloc(N_syn*N*sizeof(float));
	hipGetErrorString(hipMalloc(&d_conn_w, N_syn*N*sizeof(float)));
	printf("Memory allocated for connectivity matrix\n");

	// allocate memory on the GPU memory for the connectivity
	int *d_conn_idx, *h_conn_idx;
	h_conn_idx = (int *)malloc(N_syn*N*sizeof(int));
	hipGetErrorString(hipMalloc(&d_conn_idx, N_syn*N*sizeof(int)));

	// for synaptic input to N neurons
	float *d_Isyn;
	hipGetErrorString(hipMalloc(&d_Isyn, N*sizeof(float)));
	printf("Memory allocated for synapses\n");

	// gridSize and blockSize for N operations
	int blockSizeN;		// The launch configurator returned block size
	int minGridSize;	// The minimum grid size needed to achieve the
						// maximum occupancy for a full device launch
	int gridSizeN;		// The actual grid size needed, based on input size
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSizeN, initNeuron, 0, 0);
	// Round up according to array size
	gridSizeN = (N + blockSizeN - 1) / blockSizeN;
	
	// calculate theoretical occupancy
  	int maxActiveBlocks;
  	hipOccupancyMaxActiveBlocksPerMultiprocessor( &maxActiveBlocks, 
                                                 	initNeuron, blockSizeN, 
                                                 	0);

  	int device;
  	hipDeviceProp_t props;
  	hipGetDevice(&device);
  	hipGetDeviceProperties(&props, device);

  	float occupancy = (maxActiveBlocks * blockSizeN / props.warpSize) / 
                    		(float)(props.maxThreadsPerMultiProcessor / 
                            		props.warpSize);

  	printf("Launched blocks of size %d. Theoretical occupancy: %f\n", 
         blockSizeN, occupancy);

	
	printf("blockSizeN = %d , gridSizeN = %d, minGridSize = %d  maxActiveBlocks = %d \n", blockSizeN, gridSizeN, minGridSize, maxActiveBlocks);
	// initialize random number generator to be used for stochastic input
	printf("Initializing random number generators\n");
	hiprandState_t *devStates;
	hipGetErrorString(hipMalloc((void **)&devStates, N*sizeof(hiprandState_t)));
	
	//int sMemSize = sizeof(double) * ((THREADS_PER_BLOCK/32) + 1);
 	int sMemSize = 0;
  	int numBlocksPerSm = 0;
  	//int numThreads = THREADS_PER_BLOCK;  			
	
	hipDeviceProp_t deviceProp;
  	int devID = findCudaDevice(argc, (const char **)argv);
  	checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));
	int numSms = deviceProp.multiProcessorCount;   	
	
	int minGridSize_rng, blockSizeN_rng, gridSizeN_rng;
	checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize_rng, &blockSizeN_rng, stateupdate, 0, 0));
				
  	checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      					&numBlocksPerSm, initNeuron, blockSizeN_rng, sMemSize));
      	gridSizeN_rng = (N + blockSizeN_rng - 1) / blockSizeN_rng;
  				
  	dim3 dimGrid_rng(gridSizeN_rng, 1, 1),
      		dimBlock_rng(blockSizeN_rng, 1, 1); //
    	occupancy = (numBlocksPerSm * blockSizeN_rng / props.warpSize) / 
                 		(float)(props.maxThreadsPerMultiProcessor / 
                            		props.warpSize);
      	printf("randgeneratorinit numSms = %d  numBlocksPerSm = %d Theoretical occupancy: %f \n", numSms , numBlocksPerSm, occupancy);
      				
      	printf("randgeneratorinit blockSizeN_rng = %d , maxActiveBlocks = %d gridSizeN_rng =%d \n", blockSizeN_rng, numSms * numBlocksPerSm, gridSizeN_rng);
		
	//randgeneratorinit<<<gridSizeN, blockSizeN>>>(time(NULL), devStates);
	randgeneratorinit<<<dimGrid_rng, dimBlock_rng>>>(time(NULL), devStates, N);
	hipDeviceSynchronize();

	// Initialize connectivity matrix on the GPU
	printf("Initializing random connectivity matrix values\n");

	int idx;
	size_t postsynidx_size;
	int postsynidx[N];						// postsynaptic neuron index
	float rand_float;

	srand((unsigned) time(NULL));
	for(int i=0; i < N; i++){				// run over neurons
		postsynidx_size = N;
		for ( int j=0; j<N; j++ ){			// initialize postsynaptic idx
			postsynidx[j] = j;
		}

		for(int j=0; j<N_syn; j++){			//run over synapses
			idx = (int) rand() % postsynidx_size;
			if ( i < N_exc ){
				rand_float = (1000.0f/N_syn)* 0.5f*(float) rand()/RAND_MAX;
			}else {
				rand_float = (1000.0f/N_syn)* -1.0f*(float) rand()/RAND_MAX;
			}
			h_conn_w[i*N_syn+j] = rand_float;
			h_conn_idx[i*N_syn+j] = postsynidx[idx];
			memmove(postsynidx+idx, postsynidx+idx+1, (postsynidx_size-idx-1)*sizeof(int));
			postsynidx_size--;
		}
		if (i%1000==0)
			cout << "neuron " << i << " had connections configured.\n";
	}

	// Copy connectivity matrix to GPU memory
	printf("Retrieving initial connectivity matrix\n");
	hipGetErrorString(hipMemcpy(d_conn_w, h_conn_w, N_syn*N*sizeof(float), hipMemcpyHostToDevice));
	hipGetErrorString(hipMemcpy(d_conn_idx, h_conn_idx, N_syn*N*sizeof(int), hipMemcpyHostToDevice));
	hipDeviceSynchronize();

	// CUDA kernel initiation parameters
	// gridSize and blockSize for N operations
	int blockSizeNNsyn;		// The launch configurator returned block size
	int minGridSizeNNsyn;	// The minimum grid size needed to achieve the
							// maximum occupancy for a full device launch
	int gridSizeNNsyn;		// The actual grid size needed, based on input size
	
	/////////////////////////////////////////  S Algo //////////////////////////////////////
	hipOccupancyMaxPotentialBlockSize(&minGridSizeNNsyn, &blockSizeNNsyn, deliverspks2, 0, 0);

	// Round up according to array size
	gridSizeNNsyn = (N*N_syn + blockSizeNNsyn - 1) / blockSizeNNsyn;
		
	// run this network configuration
	// for three firing states (mode: 0 quiet, 1 balanced, 2 irregular)
	// with three algorithms (dynamic: 0 AP, 1 N, 2 S 3 AB 4 SKL)
	int d_count = atoi(argv[3]);
	for (int mode=0; mode<3; mode++){
		for (int dynamic=0; dynamic<5; dynamic=dynamic+1){

			// Initialize neural parameters for excitatory and inhibitory neurons
			printf("Initializing neuron parameters\n");
			
			int minGridSize_ini, blockSizeN_ini, gridSizeN_ini;
			checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize_ini, &blockSizeN_ini, stateupdate, 0, 0));
				
  			checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      					&numBlocksPerSm, initNeuron, blockSizeN_ini, sMemSize));
      			gridSizeN_ini = (N + blockSizeN_ini - 1) / blockSizeN_ini;
  				
  			dim3 dimGrid_ini(gridSizeN_ini, 1, 1),
      			dimBlock_ini(blockSizeN_ini, 1, 1); //
      			occupancy = (numBlocksPerSm * blockSizeN_ini / props.warpSize) / 
                    		(float)(props.maxThreadsPerMultiProcessor / 
                            		props.warpSize);
      			printf("initNeuron numSms = %d  numBlocksPerSm = %d Theoretical occupancy: %f \n", numSms , numBlocksPerSm, occupancy);
      				
      			printf("initNeuron blockSizeN_ini = %d , maxActiveBlocks = %d gridSizeN_ini =%d \n", blockSizeN_ini, numSms * numBlocksPerSm, gridSizeN_ini);
			
			//initNeuron<<<gridSizeN, blockSizeN>>>(N_exc, N, d_Neuron, devStates);
			initNeuron<<<dimGrid_ini, dimBlock_ini>>>(N_exc, N, d_Neuron, devStates);
			hipDeviceSynchronize();

			// Copy initial values of neural parameters back to CPU
			printf("Retrieving initial parameter values\n");
			hipGetErrorString(hipMemcpy(h_Neuron, d_Neuron, N*sizeof(Neuron), hipMemcpyDeviceToHost));
			hipDeviceSynchronize();

			// Print out the simulation protocol on the screen
			cout << "===================================================== \n";
			cout << "Neurons: " << N << " Synapses: " << N_syn << " Dynamic (Algorithm): " << dynamic << " Mode (State): " << mode << "\n";
			cout << "===================================================== \n";

			// Open file streams to write data
			// Make folder to save simulation results into
			char buffer[100];
			snprintf(buffer, sizeof(buffer), "Results/");
			mkdir(buffer, S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
			//snprintf(buffer, sizeof(buffer), "Results/N%dNsyn%dRegime%dAlg%d/", N, N_syn, mode, dynamic);
			snprintf(buffer, sizeof(buffer), "Results/N%dNsyn%dRegime%dAlg%d/", N, N_syn, mode, dynamic);
			mkdir(buffer, S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);

			// Write initial neural values in the file neuroninfo.csv
			string fname;
			printf("Initializing file streams to write neuron info\n");
			//fname = filename(buffer, "neuroninfo.csv");
			fname = string(buffer) +  "neuroninfo.csv";
			
			ofstream neuroninfo(fname.c_str());

			neuroninfo << "a b c d v u I id nospks" <<  "\n";
			for ( int i=0; i<N; i++ ){
				writeNeuronInfo(neuroninfo, h_Neuron[i]);
			}

			// File to write spike times and spiking neuron ids
			//fname = filename(buffer, "spiketimes.csv");
			fname = string(buffer) + "spiketimes.csv";
			
			ofstream spikes(fname.c_str());
			spikes << "SpkTime NeuronID" << "\n";

			// File to write compute time per timestep  
			//fname = filename(buffer, "computetime.csv");
			fname = string(buffer) + "computetime.csv";
			ofstream computetime(fname.c_str());
			computetime << "TimeStep SpksPerStep TimeKernelUpdate TimeSpent" << "\n";

			// initialize clocks for timekeeping
			clock_t start_sim, end_sim, start, middle, end;
			int spkcount;
			double took, took1, elapsed;
			elapsed = 0;

			cout << "\nStarting simulation and timer...\n";
			int totalspkcount = 0;
			
			unsigned int *d_totalspkcount; int *d_spkcount; // int *d_count;
			//h_spikes = (bool *)malloc(N*sizeof(bool));
			hipGetErrorString(hipMalloc(&d_totalspkcount, sizeof(unsigned int)));
			hipGetErrorString(hipMalloc(&d_spkcount, sizeof(int)));
			//hipGetErrorString(hipMalloc(&d_count, sizeof(int)));
			
			printf("blockSizeN = %d , gridSizeN = %d \n", blockSizeN, gridSizeN);
			printf("blockSizeNNsyn = %d , gridSizeNNsyn = %d \n", blockSizeNNsyn, gridSizeNNsyn);
			
			hipGetErrorString(hipMemcpy(d_totalspkcount, &totalspkcount,  sizeof(unsigned int), hipMemcpyHostToDevice));
				
			//hipGetErrorString(hipMemcpy(d_count, &spkcount, sizeof(int), hipMemcpyHostToDevice));
			hipGetErrorString(hipMemcpy(d_spkcount, &spkcount, sizeof(int), hipMemcpyHostToDevice));
			
			//stateupdate parameters for SOTA - AP N S-algo		
			int minGridSize_su, blockSizeN_su, gridSizeN_su;
			checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize_su, &blockSizeN_su, stateupdate, 0, 0));
  			checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      					&numBlocksPerSm, stateupdate, blockSizeN_su, sMemSize));
      					
      			gridSizeN_su = (N + blockSizeN_su - 1) / blockSizeN_su;
  				
  			dim3 dimGrid_su(gridSizeN_su, 1, 1),
      				dimBlock_su(blockSizeN_su, 1, 1); //
      			occupancy = (numBlocksPerSm * blockSizeN_su / props.warpSize) / 
                    		(float)(props.maxThreadsPerMultiProcessor / 
                            		props.warpSize);
      			printf("stateupdate numSms = %d  numBlocksPerSm = %d Theoretical occupancy: %f \n", numSms , numBlocksPerSm, occupancy);
      				
      			printf("stateupdate blockSizeN_su = %d , maxActiveBlocks = %d gridSizeN_su =%d \n", blockSizeN_su, numSms * numBlocksPerSm, gridSizeN_su);
      				
      			//AP-algo parameters		
      			int minGridSize_ps, blockSizeN_ps, gridSizeN_ps;
			checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize_ps, &blockSizeN_ps, propagatespikes, 0, 0));
  			checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      					&numBlocksPerSm, propagatespikes, blockSizeN_ps, sMemSize));
      			gridSizeN_ps = (N + blockSizeN_ps - 1) / blockSizeN_ps;
  				
  			dim3 dimGrid_ps(gridSizeN_ps, 1, 1),
      			dimBlock_ps(blockSizeN_ps, 1, 1); //
      			occupancy = (numBlocksPerSm * blockSizeN_ps / props.warpSize) / 
                    		(float)(props.maxThreadsPerMultiProcessor / 
                            		props.warpSize);
      			printf("0 AP-algo propagatespikes numSms = %d  numBlocksPerSm = %d Theoretical occupancy: %f \n", numSms , numBlocksPerSm, occupancy);
      				
      			printf("0 AP-algo propagatespikes blockSizeN_ps = %d , maxActiveBlocks = %d gridSizeN_ps =%d \n", blockSizeN_ps, numSms * numBlocksPerSm, gridSizeN_ps);
      			
      			//N-algo parameters		
      			int minGridSize_N, blockSizeN_N, gridSizeN_N;
			checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize_N, &blockSizeN_N, deliverspks1, 0, 0));
  			checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      					&numBlocksPerSm, deliverspks1, blockSizeN_N, sMemSize));
      			gridSizeN_N = (N + blockSizeN_N - 1) / blockSizeN_N;
  			  			
  			dim3 dimGrid_N(gridSizeN_N, 1, 1),
      			dimBlock_N(blockSizeN_N, 1, 1); //
      			occupancy = (numBlocksPerSm * blockSizeN_N / props.warpSize) / 
                    		(float)(props.maxThreadsPerMultiProcessor / 
                            		props.warpSize);
      			printf("1 N-algo deliverspks1 numSms = %d  numBlocksPerSm = %d Theoretical occupancy: %f \n", numSms , numBlocksPerSm, occupancy);
      			printf("1 N-algo deliverspks1 blockSizeN_N = %d , maxActiveBlocks = %d gridSizeN_N =%d \n", blockSizeN_N, numSms * numBlocksPerSm, gridSizeN_N);
      			
      			/// S-algo parameters	
      			int minGridSize_N2, blockSizeN_N2, gridSizeN_N2;
			checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize_N2, &blockSizeN_N2, deliverspks2, 0, 0));
  			checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      					&numBlocksPerSm, deliverspks2, blockSizeN_N2, sMemSize));
      			gridSizeN_N2 = (N*N_syn + blockSizeN_N2 - 1) / blockSizeN_N2;  //gridSizeNNsyn = (N*N_syn + blockSizeNNsyn - 1) / blockSizeNNsyn;
  				
  			dim3 dimGrid_N2(gridSizeN_N2, 1, 1),
      				dimBlock_N2(blockSizeN_N2, 1, 1); //
      			occupancy = (numBlocksPerSm * blockSizeN_N2 / props.warpSize) / 
                    		(float)(props.maxThreadsPerMultiProcessor / 
                            		props.warpSize);
      			printf("2 S-algo deliverspks2 numSms = %d  numBlocksPerSm = %d Theoretical occupancy: %f \n", numSms , numBlocksPerSm, occupancy);
      			printf("2 S-algo deliverspks2 blockSizeN_N2 = %d , maxActiveBlocks = %d gridSizeN_N2 =%d \n", blockSizeN_N2, numSms * numBlocksPerSm, gridSizeN_N2);
      			
      			///// state update parameters for AB-algo using persistence
      			int minGridSize_per, blockSizeN_per, gridSizeN_per;
			checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize_per, &blockSizeN_per, AB_stateupdate, 0, 0));				
  			checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      					&numBlocksPerSm, AB_stateupdate, blockSizeN_per, sMemSize));
      			gridSizeN_per = numSms * numBlocksPerSm;
  			dim3 dimGrid_per(gridSizeN_per, 1, 1),
      				dimBlock_per(blockSizeN_per, 1, 1); //
      			occupancy = (numBlocksPerSm * blockSizeN_per / props.warpSize) / 
                    		(float)(props.maxThreadsPerMultiProcessor / 
                            		props.warpSize);
      			printf("3 AB_stateupdate numSms = %d  numBlocksPerSm = %d Theoretical occupancy: %f \n", numSms , numBlocksPerSm, occupancy);
      			printf("3 AB_stateupdate blockSizeN_per = %d , maxActiveBlocks = %d gridSizeN_per =%d \n", blockSizeN_per, numSms * numBlocksPerSm, gridSizeN_per);  
      			void *kernelArgs_per[] = {
     				 (void *)&d_Neuron,  (void *)&d_spikes, (void *)&d_Isyn, (void *)&devStates, (void *)&N, (void *)&N_exc, (void *)&mode,};
      				
      			///// spike propagation parameters for AB-algo using persistence	
      			int minGridSize_N3, blockSizeN_N3, gridSizeN_N3;
			checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize_N3, &blockSizeN_N3, AB_deliverspks, 0, 0));
  				checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      					&numBlocksPerSm, AB_deliverspks, blockSizeN_N3, sMemSize));
      				//gridSizeN_N2 = (N*N_syn + blockSizeN_N2 - 1) / blockSizeN_N2;  //gridSizeNNsyn = (N*N_syn + blockSizeNNsyn - 1) / blockSizeNNsyn;
  			gridSizeN_N3 = numSms * numBlocksPerSm;
  			dim3 AB_dimGrid(gridSizeN_N3, 1, 1),
      				AB_dimBlock(blockSizeN_N3, 1, 1); //
      			occupancy = (numBlocksPerSm * blockSizeN_N3 / props.warpSize) / 
                    		(float)(props.maxThreadsPerMultiProcessor / 
                            		props.warpSize);
      			printf("3 AB AB_deliverspks numSms = %d  numBlocksPerSm = %d Theoretical occupancy: %f \n", numSms , numBlocksPerSm, occupancy);
      			printf("3 AB AB_deliverspks blockSizeN_N3 = %d , maxActiveBlocks = %d gridSizeN_N3 =%d \n", blockSizeN_N3, numSms * numBlocksPerSm, gridSizeN_N3);
      			      			
      			void *AB_kernelArgs[] = {
     				 (void *)&d_Neuron,  (void *)&d_spikes, (void *)&d_conn_w, (void *)&d_conn_idx,
      				(void *)&d_Isyn, (void *)&N, (void *)&N_exc, (void *)&N_inh, (void *)&N_syn, };
			
			/// SKL-algo	parameters	 stateupdate included in the kernel
			int minGridSize_de, blockSizeN_de, gridSizeN_de;
			checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize_de, &blockSizeN_de, SKL_deliverspks, 0, 0));				
  			checkCudaErrors(hipOccupancyMaxActiveBlocksPerMultiprocessor(
      					&numBlocksPerSm, SKL_deliverspks, blockSizeN_de, sMemSize));
      			gridSizeN_de = numSms * numBlocksPerSm;
  			dim3 SKL_dimGrid(gridSizeN_de, 1, 1),
      				SKL_dimBlock(blockSizeN_de, 1, 1); //
      			occupancy = (numBlocksPerSm * blockSizeN_de / props.warpSize) / 
                    		(float)(props.maxThreadsPerMultiProcessor / 
                            		props.warpSize);
      			printf("4 SKL SKL_deliverspks numSms = %d  numBlocksPerSm = %d Theoretical occupancy: %f \n", numSms , numBlocksPerSm, occupancy);
      			printf("4 SKL SKL_deliverspks blockSizeN_de = %d , maxActiveBlocks = %d gridSizeN_de =%d \n", blockSizeN_de, numSms * numBlocksPerSm, gridSizeN_de);
							
      			void *SKL_kernelArgs[] = {
     				 (void *)&d_Neuron,  (void *)&d_spikes, (void *)&d_conn_w, (void *)&d_conn_idx,
      				(void *)&d_Isyn, (void *)&N, (void *)&N_exc, (void *)&N_syn, (void *)&d_totalspkcount, (void *)&devStates, (void *)&mode, (void *)&d_count, };    			
			/////////////////////	//////////////////////////////////////////////////////////////
			start_sim = clock();
			
			hipProfilerStart();		
			
			if (!(dynamic ==4))
			{

			  // Start main simulation loop  ITER_COUNT
			  for (int tstep=0; tstep<d_count; tstep++){

				spkcount = 0;
				//hipDeviceSynchronize();
				start = clock();
				// State update
				
				if ((dynamic ==4 ) ) 
					hipLaunchCooperativeKernel((void *)AB_stateupdate, dimGrid_per, dimBlock_per, kernelArgs_per, sMemSize, NULL);	//for AB-algo		
						//AB_stateupdate<<<dimGrid_per, dimBlock_per>>>(d_Neuron, d_spikes, d_Isyn, devStates, N, N_exc, mode); 
				else 
					stateupdate<<<dimGrid_su, dimBlock_su>>>(d_Neuron, d_spikes, d_conn_w, d_conn_idx, d_Isyn, devStates, N, N_exc, N_syn, gridSizeN_ps, blockSizeN_ps, dynamic, mode); // for SOTA 
				
				middle = clock();
				
				switch ( dynamic ){
				case 0:
					// if dynamic parallelization, stateupdate kernel already calculated  AP-algorithm
					break;
				case 1:
					// parallelization over neurons: N-algorithm
					//hipDeviceSynchronize();
					deliverspks1<<<dimGrid_N, dimBlock_N>>> (d_Neuron, d_spikes, d_conn_w, d_conn_idx, d_Isyn, N, N_exc, N_inh, N_syn);
					break;
				case 2:
					// parallelization over synapses: S-algorithm
					//hipDeviceSynchronize();
					deliverspks2<<<dimGrid_N2, dimBlock_N2>>> (d_Neuron, d_spikes, d_conn_w, d_conn_idx, d_Isyn, N, N_exc, N_inh, N_syn);
					break;
					
				case 3:
					// parallelization over synapses: AB-algorithm     Modified S-algorithm using persistence   
					//AB_deliverspks<<<AB_dimGrid, AB_dimBlock>>> (d_Neuron, d_spikes, d_conn_w, d_conn_idx, d_Isyn, N, N_exc, N_inh, N_syn, gridSizeNNsyn);
					hipLaunchCooperativeKernel((void *)AB_deliverspks,
                                              AB_dimGrid, AB_dimBlock, AB_kernelArgs,
                                              sMemSize, NULL);				
				}
				//hipDeviceSynchronize();
				end = clock();
				hipMemcpy(h_spikes, d_spikes, N*sizeof(bool), hipMemcpyDeviceToHost);
				//hipDeviceSynchronize();
				for ( int i=0; i<N; i++){
					if ( h_spikes[i] == true ){
						spikes << tstep << " " << i << "\n";
						spkcount++;
					}
				}

				took1 = double(end-middle) / CLOCKS_PER_SEC * 1000;
				took = double(end-start) / CLOCKS_PER_SEC * 1000;
				elapsed += took;

				computetime << tstep << " " << spkcount << " " << took1 << " " << took << "\n";
				//printf("spkcount =%d  totalspkcount =%d\t", spkcount, totalspkcount);
				totalspkcount += spkcount;

			  }
			  end_sim = clock();
			}
			
			else
			{
				start_sim = clock();				
				if(dynamic ==4) {	  //dynamic ==4    SKL-algorithm                               	
                               	totalspkcount = 0;	// *totalspkcount_new = 0; 
                               	elapsed = 0;	
                               	start = clock();                               	
                               	hipLaunchCooperativeKernel((void *)SKL_deliverspks,
                                              SKL_dimGrid, SKL_dimBlock, SKL_kernelArgs,
                                              sMemSize, NULL);
                                      end = clock();
                                      //hipMemcpy(&totalspkcount, d_totalspkcount, sizeof(unsigned int), hipMemcpyDeviceToHost);
                                      took1 = double(end-start) / CLOCKS_PER_SEC * 1000;
					took = double(end-start) / CLOCKS_PER_SEC * 1000;
					elapsed += took;
			   		//totalspkcount = (int)*totalspkcount_new;	
			   		//end_sim = clock();	
                                 }
                                   hipMemcpy(&totalspkcount, d_totalspkcount, sizeof(unsigned int), hipMemcpyDeviceToHost); //d_spkcount
				    end_sim = clock();			//totalspkcount = spkcount;	
			  	//hipDeviceSynchronize();	
				
				//computetime << tstep << " " << spkcount << " " << took1 << " " << took << "\n";
				//totalspkcount += spkcount;			   
			   
			}
			//end_sim = clock();			
			hipProfilerStop();
			cout << "\nClocks per sec is " << CLOCKS_PER_SEC << "\n";
			cout << "\nEnd of simulation...\n";
			cout << "Simulation took: " << double(end_sim-start_sim) / CLOCKS_PER_SEC * 1000<< " ms.\n";
			cout << "Without data transfers: " << elapsed << " ms.\n";
			cout << "Total number of spikes: " << totalspkcount << "\n\n\n";

			// Write simulation overview in the file sim_overview.csv
			//ofstream sim_overview(filename(buffer, "sim_overview.csv"));
			ofstream sim_overview(string(buffer) + "sim_overview.csv");
			sim_overview << "N elapsed(ms) elapsedwithdata(ms) totalspkcount" <<  "\n";
			sim_overview << N << " " << elapsed << " " << double(end_sim-start_sim) / CLOCKS_PER_SEC * 1000 << " " << totalspkcount;
                       //sim_overview.close();

			// Write neural values in the file neuroninfo.csv
			//ofstream neuroninfoafter(filename(buffer, "neuroninfoafter.csv"));
			ofstream neuroninfoafter(string(buffer) + "neuroninfoafter.csv");
			if ( ! neuroninfoafter.is_open() ){
				//std::cout <<  buffer << std::endl;
    				std::cerr << "Error! Failed to open file "  << std::endl;
    				return 1;
			}
			neuroninfoafter << "a b c d v u I id nospks" <<  "\n";
			for ( int i=0; i<N; i++ ){
				writeNeuronInfo(neuroninfoafter, h_Neuron[i]);
			}
			//neuroninfoafter.close();
		}
	}
	hipDeviceReset();	
}
